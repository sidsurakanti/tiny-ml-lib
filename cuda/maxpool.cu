#include "hip/hip_runtime.h"
#include "errors.cuh"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_math_constants.h>

struct nchw {
  int n, c, w, h;
};

void printMat(float *mat, nchw size) {
  auto [n, c, w, h] = size;
  printf("\n");
  for (int i = 0; i < (n * c * w * h); i++) {
    printf("%6.2f ", mat[i]);
    if ((i + 1) % (w * h) == 0)
      printf("\n");
    if ((i + 1) % w == 0)
      printf("\n");
    else if ((i + 1) % (c * w * h) == 0)
      printf("\n");
  }
}

void fillMat(float *mat, int size) {
  srand(3333);
  for (int i = 0; i < size; i++) {
    mat[i] = ((float)(rand() % 10));
  }
}

template <int kernel_size = 2>
// N, C, H, W
__global__ void MaxPoolKernel(float *in, float *out, int N, int C, int H, int W,
                              int outH, int outW, int stride = 2) {
  // assume we use only 2d blocks
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // 3d grid (N, H, W)
  int bz = blockIdx.z; // n (batch i)
  int by = blockIdx.y;
  int bx = blockIdx.x;

  // works for 1 channel
  float buffer[kernel_size][kernel_size];

  int batch = bz * blockDim.z; // assume tz == 0 (n)
  // row, col for out matrix
  int outRow = ty + by * blockDim.y;
  int outCol = tx + bx * blockDim.x;

  if (outRow >= outH || outCol >= outW)
    return;

  int rowStart = outRow * stride;
  int colStart = outCol * stride;

  // processing each channel is very small amount of work for each thread
  // so we can make it work on all the channels (reasonably < 512)
  for (int c = 0; c < C; c++) {
    // load kernel into buffer
    for (int i = 0; i < kernel_size; i++) {
      for (int j = 0; j < kernel_size; j++) {
        int row = rowStart + i;
        int col = colStart + j;
        buffer[i][j] = in[((batch * C * H * W) + (c * H * W)) + row * W + col];
      }
    }

    // apply maxpool on the buffer
    float max = -HIP_INF;
    for (int i = 0; i < kernel_size; i++) {
      for (int j = 0; j < kernel_size; j++) {
        max = fmaxf(buffer[i][j], max);
      }
    }

    // printf("%f", max);

    // add res to output buf
    out[((batch * C * outH * outW) + (c * outH * outW)) + (outRow * outW) +
        outCol] = max;
  }
}

int main() {
  std::cout << "launching gpu kernel..." << std::endl;

  // N, H, W (we're gonna just put in C in the kernel)
  nchw dimsIn{2, 2, 4, 4};
  int sizeIn = dimsIn.n * dimsIn.c * dimsIn.h * dimsIn.w;
  int memsizeIn = sizeIn * sizeof(float);
  float *in_h;

  CU_CHECK(hipHostMalloc(&in_h, memsizeIn));

  nchw dimsOut{dimsIn.n, dimsIn.c, 2, 2};
  int sizeOut = dimsOut.n * dimsOut.c * dimsOut.h * dimsOut.w;
  int memsizeOut = sizeOut * sizeof(float);
  float *out_h;
  CU_CHECK(hipHostMalloc(&out_h, memsizeOut));

  fillMat(in_h, sizeIn);
  fillMat(out_h, sizeOut);

  int BLOCK_SIZE = 16;
  int gridRows = (dimsOut.h + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int gridCols = (dimsOut.w + BLOCK_SIZE - 1) / BLOCK_SIZE;
  // n, ceil(out_h) / block_size, same for w
  dim3 gridDim(gridCols, gridRows, dimsOut.n); // blocks in grid
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);       // threads in block

  float *in_d, *out_d;
  hipMalloc(&in_d, memsizeIn);
  hipMalloc(&out_d, memsizeOut);
  hipMemcpy(in_d, in_h, memsizeIn, hipMemcpyHostToDevice);

  MaxPoolKernel<<<gridDim, blockDim>>>(in_d, out_d, dimsIn.n, dimsIn.c,
                                       dimsIn.h, dimsIn.w, dimsOut.h,
                                       dimsOut.w);
  CU_CHECK(hipDeviceSynchronize());

  hipMemcpy(out_h, out_d, memsizeOut, hipMemcpyDeviceToHost);

  printf("in shape (%d, %d, %d, %d)\n", dimsIn.n, dimsIn.c, dimsIn.h, dimsIn.w);
  printf("out shape (%d, %d, %d, %d)\n", dimsOut.n, dimsOut.c, dimsOut.h,
         dimsOut.w);
  printMat(in_h, dimsIn);
  printf("\n");
  printMat(out_h, dimsOut);

  std::cout << "job finished!" << std::endl;
  return 0;
}
