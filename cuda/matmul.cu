#include "hip/hip_runtime.h"
#include "errors.cuh"
#include "matmul.cuh"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdexcept>

// ** lowk this file has a lot of dumb comments but that's js me thinking
// ** feel free to ignore it it's a whole yap city down there
// also prolly the messiest file i've ever written

// BIG IDEA:
// set up matrices
// cuda malloc on device
// copy matrices from host -> device
// do matmul
// cpy back to host

const bool DEBUG = false;
const bool CPU = false;
const int BLOCK_SIZE = 16;

void matInit(float *mat, int size, int n) {
  for (int i = 0; i < size; i++) {
    mat[i] = n;
  }
}

void printMat(float *mat, int size, int colWidth) {
  for (int i = 0; i < size; i++) {
    printf("%6.2f ", mat[i]);
    if ((i + 1) % colWidth == 0)
      printf("\n");
  }
}

void cpuMatMul(float *A, float *B, float *C, int m, int n, int k) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < k; col++) {
      for (int i = 0; i < n; i++) {
        C[row * k + col] += A[row * n + i] * B[k * i + col];
      }
    }
  }
}

__global__ void ReluKernel(float *mat, int m, int n) {
  int row = threadIdx.y + (blockIdx.y * blockDim.y);
  int col = threadIdx.x + (blockIdx.x * blockDim.x);

  if (row >= m || col >= n)
    return;

  float &value = mat[row * n + col];
  value = value > 0 ? value : 0;
}

__global__ void VecMatAddKernel(float *vec, float *mat, int m, int n) {
  int row = threadIdx.y + (blockIdx.y * blockDim.y);
  int col = threadIdx.x + (blockIdx.x * blockDim.x);

  if (row >= m || col >= n)
    return;

  mat[row * n + col] += vec[col];
}

// make sure vec and mat are 1d row majored otherwise you're cooked bro
void vecMatAdd(float *vec, float *mat, int m, int n, int k) {
  if (k != m)
    throw std::runtime_error("Dims mismatch, rows of vec and matrix have to be "
                             "equal (you're ngmi if you keep doing ts bruv "
                             "come back with proper inputs)");
  dim3 blockDim(32, 32);
  dim3 gridDim((m + blockDim.x - 1) / blockDim.x,
               (n + blockDim.y) / blockDim.y);

  VecMatAddKernel<<<gridDim, blockDim>>>(vec, mat, m, n);
}

__global__ void BasicMatMulKernel(float *A, float *B, float *C, int m, int n,
                                  int k) {
  // NOTE: y for rows (vertical) in cuda and x (horiz) for cols
  int offsetY = blockDim.y * blockIdx.y;
  int offsetX = blockDim.x * blockIdx.x;

  // NOTE:
  // >>> think each thread == 1 element of C that we need to calculate
  // >>> so, using threadIdx.x and .y we can identify the row and col to
  // multiply for that specific element
  // >>> considering there's gonna be multiple
  // blocks per grid, we'd have to add the offset to get the actual global
  // threadIdx
  int row = threadIdx.y + offsetY;
  int col = threadIdx.x + offsetX;

  // bounds checking so we don't access uninit'd memory
  if (row >= m || col >= k)
    return;

  float sum = 0.0f;
  // NOTE: have to index by row major bc everything is 1D
  for (int i = 0; i < n; i++) {
    sum += A[row * n + i] * B[k * i + col];
  }

  C[row * k + col] = sum;
}

// (m, n) * (n, k) = (m, k)
template <int block_size>
__global__ void MatMulKernel(float *A, float *B, float *C, int m, int n,
                             int k) {
  int offsetY = blockDim.y * blockIdx.y;
  int offsetX = blockDim.x * blockIdx.x;
  int row = threadIdx.y + offsetY;
  int col = threadIdx.x + offsetX;

  // clang-format off
  //
  // NOTE: BIG IDEA:
  // we load partial 16x16 tiles of A and B into shared memory for each element of C contained in the 16x16 thread block; 
  // then we perform partial mat mul;
  // we keep doing this until we can iterate thru all (n) rows in A and (n) cols in B; 
  // at which point we'll have fully completed the matrix mul for a 16x16 block of C;
  //
  // ===========
  // PSUEDO CODE # i dont think anyone except me can understand the bullshit below this line
  //
  // tile iters = ceil(n/blocksize)
  //
  // each thread (an element of C) of the 16x16 block loads
  // their row/col to shared mem: 
  // a[1d row idx + (t * block_size + tx)] & 
  // b[k * (t * block_size + ty) + col]
  //
  // sync the threads so that both BLOCKSIZE*BLOCKSIZE blocks of A & B are
  // loaded in shared mem for the SM;
  //
  // perform partial matrix mul
  // tileA[ty][k:0->15] tileB[k:0->15][tx]!
  //
  // add to accum
  // sync threads again

  __shared__ float tileA[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float tileB[BLOCK_SIZE][BLOCK_SIZE];

  int iters = (n + block_size - 1) / block_size;
  int ty = threadIdx.y;
  int tx = threadIdx.x;

  float sum = 0.0f;

  for (int t = 0; t < iters; t++) {
    // check if the element we're accessing is out of bounds in the matrix;
    if ((block_size * t + tx < n) && (row < m))
      tileA[ty][tx] = A[row * n + (t * block_size + tx)];
    else 
      tileA[ty][tx] = 0.0f;

    if ((block_size * t + ty < n) && (col < k)) 
      tileB[ty][tx] = B[k * (t * block_size + ty) + col];
    else 
      tileB[ty][tx] = 0.0f;

    __syncthreads();

    // perform mat mul
    for (int i = 0; i < BLOCK_SIZE; i++) {
      sum += tileA[ty][i] * tileB[i][tx];
    }

    // sync threads before we pull next tile into shared memory
    __syncthreads();
  }

  // bounds check here also
  if (row < m && col < k) 
    C[row * k + col] = sum;
}


void matMul() {
  // SET UP DATA

  // m, n (A) && n, k (B)
  // dim3 dimsA(8, 5);
  // dim3 dimsB(5, 10);
  // dim3 dimsA(2048, 2048);
  // dim3 dimsB(2048, 2048);
  dim3 dimsA(16384, 8192);
  dim3 dimsB(8192, 32768);
  dim3 dimsC(dimsA.x, dimsB.y);

  printf("[OPERATION] (%d, %d) * (%d, %d)\n", dimsA.x, dimsA.y, dimsB.x,
         dimsB.y);

  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int size_C = dimsC.x * dimsC.y;

  unsigned int mem_sizeA = sizeof(float) * size_A;
  unsigned int mem_sizeB = sizeof(float) * size_B;
  unsigned int mem_sizeC = sizeof(float) * size_C;

  float *A_h, *A_d, *B_h, *B_d, *C_h, *C_d;

  // alloc pinned memory on host for faster cpy times
  CU_CHECK(hipHostMalloc(&A_h, mem_sizeA));
  CU_CHECK(hipHostMalloc(&B_h, mem_sizeB));
  CU_CHECK(hipHostMalloc(&C_h, mem_sizeC));

  matInit(A_h, size_A, 1);
  matInit(B_h, size_B, 2);

  // allocate device mem
  CU_CHECK(hipMalloc(&A_d, mem_sizeA));
  CU_CHECK(hipMalloc(&B_d, mem_sizeB));
  CU_CHECK(hipMalloc(&C_d, mem_sizeC));

  // copy matx's from host to device async
  // NOTE: seperate from host thread, so basically making this nonblocking
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  CU_CHECK(
      hipMemcpyAsync(A_d, A_h, mem_sizeA, hipMemcpyHostToDevice, stream));
  CU_CHECK(
      hipMemcpyAsync(B_d, B_h, mem_sizeB, hipMemcpyHostToDevice, stream));

  unsigned int gridRows = (dimsC.x + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int gridCols = (dimsC.y + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridSize(gridCols, gridRows); // blocks per grid
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE); // threads per block

  printf("[WARMING UP GPU]\n");
  // warm up runs 
  BasicMatMulKernel<<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, dimsA.x, dimsA.y, dimsB.y);
  MatMulKernel<BLOCK_SIZE><<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, dimsA.x, dimsA.y, dimsB.y);

  CU_CHECK(hipStreamSynchronize(stream));

  hipEvent_t start, stop;
  float elapsed;
  CU_CHECK(hipEventCreate(&start));
  CU_CHECK(hipEventCreate(&stop));

  printf("[RUNNING] SlowMatMul\n");
  CU_CHECK(hipEventRecord(start, stream));

  // <<<blocks in grid, block size (threads in block), dynamic shared mem,
  // gpu stream to run on>>>
  // NOTE: m = dimsA.x, n = dimsA.y, k = dimsB.y
  BasicMatMulKernel<<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, dimsA.x, dimsA.y, dimsB.y);

  CU_CHECK(hipEventRecord(stop, stream));
  CU_CHECK(hipEventSynchronize(stop));
  CU_CHECK(hipEventElapsedTime(&elapsed, start, stop));
  printf("[TIME] SlowMatMul completed in %.2fms.\n", elapsed);


  printf("[RUNNING] FastMatMul\n");
  CU_CHECK(hipEventRecord(start, stream));

  // <<<blocks in grid, block size (threads in block), dynamic shared mem,
  // gpu stream to run on>>>
  // NOTE: m = dimsA.x, n = dimsA.y, k = dimsB.y
  MatMulKernel<BLOCK_SIZE><<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, dimsA.x, dimsA.y, dimsB.y);

  CU_CHECK(hipEventRecord(stop, stream));
  CU_CHECK(hipEventSynchronize(stop));
  CU_CHECK(hipEventElapsedTime(&elapsed, start, stop));
  printf("[TIME] FastMatMul completed in %.2fms.\n", elapsed);

  CU_CHECK(hipEventDestroy(start));
  CU_CHECK(hipEventDestroy(stop));

  // wait host thread & error check
  CU_CHECK(hipGetLastError());
  CU_CHECK(hipStreamSynchronize(stream));

  // copy result back to host
  CU_CHECK(
      hipMemcpyAsync(C_h, C_d, mem_sizeC, hipMemcpyDeviceToHost, stream));
  CU_CHECK(hipStreamSynchronize(stream));
  
  // free device memory
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  hipStreamDestroy(stream);

  if (DEBUG) {
    printf("\n[MAT A] (%dx%d):\n", dimsA.x, dimsA.y);
    printMat(A_h, size_A, dimsA.y);

    printf("\n[MAT B] (%dx%d):\n", dimsB.x, dimsB.y);
    printMat(B_h, size_B, dimsB.y);

    printf("\n[MAT C = A * B] (%dx%d):\n", dimsC.x, dimsC.y);
    printMat(C_h, size_C, dimsC.y);

  }


  if (CPU) {
    float* C = (float*)calloc(size_C, sizeof(float));

    printf("[RUNNING] CpuMatMul\n");

    auto s = std::chrono::high_resolution_clock::now();
    cpuMatMul(A_h, B_h, C, dimsA.x, dimsA.y, dimsB.y);
    auto e = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(e - s);
    printf("[TIME] CPU finished in %ldms.\n", duration.count());

    if (DEBUG) printMat(C, size_C, dimsC.y);

    free(C);
  }

  // free cpu memory
  hipHostFree(C_h);
  hipHostFree(A_h);
  hipHostFree(B_h);
}

int main() {
  std::cout << "[CUDA] Launching matrix multiplication kernel...\n";

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  std::cout << "[DEVICE INFO]" << std::endl;
  std::cout << "DEVICE: " << prop.name << std::endl;
  std::cout << "SM COUNT: " << prop.multiProcessorCount << std::endl;
  std::cout << "MAX THREADS / BLOCK: " << prop.maxThreadsPerBlock << std::endl;
  std::cout << "WARP SIZE: " << prop.warpSize << std::endl;
  std::cout << "SHARED MEMORY / BLOCK: " << prop.sharedMemPerBlock / (1024) << "KB" << std::endl;
  std::cout << "GLOBAL MEMORY: " << prop.totalGlobalMem / (1024*1024*1024) << "GB" << std::endl;

  matMul();

  std::cout << "[END]" << std::endl;
  return 0;
}

