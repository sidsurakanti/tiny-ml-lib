#include "hip/hip_runtime.h"
#include "errors.cuh"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/pytypes.h>

namespace py = pybind11;

// ** lowk this file has a lot of dumb comments but that's js me thinking
// ** feel free to ignore it it's a whole yap city down there
// also prolly the messiest file i've ever written

// BIG IDEA:
// set up matrices
// cuda malloc on device
// copy matrices from host -> device
// do matmul
// cpy back to host

const bool DEBUG = false;
const int BLOCK_SIZE = 16;
const bool CPU = false;

void matInit(float *mat, int size, int n) {
  for (int i = 0; i < size; i++) {
    mat[i] = n;
  }
}

void printMat(float *mat, int size, int colWidth) {
  for (int i = 0; i < size; i++) {
    printf("%6.2f ", mat[i]);
    if ((i + 1) % colWidth == 0)
      printf("\n");
  }
}

void cpuMatMul(float *A, float *B, float *C, int m, int n, int k) {
  for (int row = 0; row < m; row++) {
    for (int col = 0; col < k; col++) {
      for (int i = 0; i < n; i++) {
        C[row * k + col] += A[row * n + i] * B[k * i + col];
      }
    }
  }
}

__global__ void SlowMatMulKernel(float *A, float *B, float *C, int m, int n,
                                 int k) {
  // NOTE: y for rows (vertical) in cuda and x (horiz) for cols
  int offsetY = blockDim.y * blockIdx.y;
  int offsetX = blockDim.x * blockIdx.x;

  // NOTE:
  // >>> think each thread == 1 element of C that we need to calculate
  // >>> so, using threadIdx.x and .y we can identify the row and col to
  // multiply for that specific element
  // >>> considering there's gonna be multiple
  // blocks per grid, we'd have to add the offset to get the actual global
  // threadIdx
  int row = threadIdx.y + offsetY;
  int col = threadIdx.x + offsetX;

  // bounds checking so we don't access uninit'd memory
  if (row >= m || col >= k)
    return;

  float sum = 0.0f;
  // NOTE: have to index by row major bc everything is 1D
  for (int i = 0; i < n; i++) {
    sum += A[row * n + i] * B[k * i + col];
  }

  C[row * k + col] = sum;
}

template <int block_size>
// (m, n) * (n, k) = (m, k)
__global__ void MatMulKernel(float *A, float *B, float *C, int m, int n,
                             int k) {
  int offsetY = blockDim.y * blockIdx.y;
  int offsetX = blockDim.x * blockIdx.x;
  int row = threadIdx.y + offsetY;
  int col = threadIdx.x + offsetX;

  // clang-format off
  //
  // NOTE: BIG IDEA:
  // we load partial 16x16 tiles of A and B into shared memory for each element of C contained in the 16x16 thread block; 
  // then we perform partial mat mul;
  // we keep doing this until we can iterate thru all (n) rows in A and (n) cols in B; 
  // at which point we'll have fully completed the matrix mul for a 16x16 block of C;
  //
  // ===========
  // PSUEDO CODE # i dont think anyone except me can understand the bullshit below this line
  //
  // tile iters = ceil(n/blocksize)
  //
  // each thread (an element of C) of the 16x16 block loads
  // their row/col to shared mem: 
  // a[1d row idx + (t * block_size + tx)] & 
  // b[k * (t * block_size + ty) + col]
  //
  // sync the threads so that both BLOCKSIZE*BLOCKSIZE blocks of A & B are
  // loaded in shared mem for the SM;
  //
  // perform partial matrix mul
  // tileA[ty][k:0->15] tileB[k:0->15][tx]!
  //
  // add to accum
  // sync threads again

  __shared__ float tileA[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float tileB[BLOCK_SIZE][BLOCK_SIZE];

  int iters = (n + block_size - 1) / block_size;
  int ty = threadIdx.y;
  int tx = threadIdx.x;

  float sum = 0.0f;

  for (int t = 0; t < iters; t++) {
    // check if the element we're accessing is out of bounds in the matrix;
    if ((block_size * t + tx < n) && (row < m))
      tileA[ty][tx] = A[row * n + (t * block_size + tx)];
    else 
      tileA[ty][tx] = 0.0f;

    if ((block_size * t + ty < n) && (col < k)) 
      tileB[ty][tx] = B[k * (t * block_size + ty) + col];
    else 
      tileB[ty][tx] = 0.0f;

    __syncthreads();

    // perform mat mul
    for (int i = 0; i < BLOCK_SIZE; i++) {
      sum += tileA[ty][i] * tileB[i][tx];
    }

    // sync threads before we pull next tile into shared memory
    __syncthreads();
  }

  // bounds check here also
  if (row < m && col < k) 
    C[row * k + col] = sum;
}

typedef typename py::array_t<float, py::array::c_style | py::array::forcecast> py_ndarray_t;

py::array matMulNp(
  py_ndarray_t A,
  py_ndarray_t B,
  int m, int n, int k
) {
  unsigned int size_A = m * n;
  unsigned int size_B = n * k;
  unsigned int size_C = m * k;

  unsigned int mem_sizeA = sizeof(float) * size_A;
  unsigned int mem_sizeB = sizeof(float) * size_B;
  unsigned int mem_sizeC = sizeof(float) * size_C;

  // unchecked is better than arr.request() for buf & then buf.ptr() 
  // cus it auto throws when given more than 1d
  const float* A_h = A.unchecked<1>().data(0); // ptr to A[0]
  const float* B_h = B.unchecked<1>().data(0); 
  float* C_h = (float*)calloc(m * k, sizeof(float));

  float* A_d, *B_d, *C_d;

  CU_CHECK(hipMalloc(&A_d, mem_sizeA));
  CU_CHECK(hipMalloc(&B_d, mem_sizeB));
  CU_CHECK(hipMalloc(&C_d, mem_sizeC));

  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  CU_CHECK(
      hipMemcpyAsync(A_d, A_h, mem_sizeA, hipMemcpyHostToDevice, stream));
  CU_CHECK(
      hipMemcpyAsync(B_d, B_h, mem_sizeB, hipMemcpyHostToDevice, stream));

  unsigned int gridRows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int gridCols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridSize(gridCols, gridRows); // blocks per grid
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE); // threads per block

  MatMulKernel<BLOCK_SIZE><<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, m, n, k);

  // wait host thread & error check
  CU_CHECK(hipGetLastError());
  CU_CHECK(hipStreamSynchronize(stream));

  // copy result back to host
  CU_CHECK(
      hipMemcpyAsync(C_h, C_d, mem_sizeC, hipMemcpyDeviceToHost, stream));
  CU_CHECK(hipStreamSynchronize(stream));

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  hipStreamDestroy(stream);

  // wrap the c ptr in a capsule with a destructor so we can safely pass it around
  // we need to do this so when python destruct's its capsule obj we auto free the memory for the c ptr
  py::capsule free_when_done(C_h, [](void *ptr) { free(ptr); });
  py::array_t<float> result = py::array_t<float>(
    {m, k}, // shape
    C_h, // data ptr
    free_when_done
  );

  return result;

}

void matMul() {
  // SET UP DATA

  // m, n
  // n, k
  // dim3 dimsA(8, 5);
  // dim3 dimsB(5, 10);
  // dim3 dimsA(2048, 2048);
  // dim3 dimsB(2048, 2048);
  dim3 dimsA(16384, 8192);
  dim3 dimsB(8192, 32768);
  dim3 dimsC(dimsA.x, dimsB.y);

  printf("[OPERATION] (%d, %d) * (%d, %d)\n", dimsA.x, dimsA.y, dimsA.y,
         dimsB.y);

  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int size_C = dimsC.x * dimsC.y;

  unsigned int mem_sizeA = sizeof(float) * size_A;
  unsigned int mem_sizeB = sizeof(float) * size_B;
  unsigned int mem_sizeC = sizeof(float) * size_C;

  float *A_h, *A_d, *B_h, *B_d, *C_h, *C_d;

  // alloc pinned memory on host for faster cpy times
  CU_CHECK(hipHostMalloc(&A_h, mem_sizeA));
  CU_CHECK(hipHostMalloc(&B_h, mem_sizeB));
  CU_CHECK(hipHostMalloc(&C_h, mem_sizeC));

  matInit(A_h, size_A, 1);
  matInit(B_h, size_B, 2);

  // allocate device mem
  CU_CHECK(hipMalloc(&A_d, mem_sizeA));
  CU_CHECK(hipMalloc(&B_d, mem_sizeB));
  CU_CHECK(hipMalloc(&C_d, mem_sizeC));

  // copy matx's from host to device async
  // NOTE: seperate from host thread, so basically making this nonblocking
  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  CU_CHECK(
      hipMemcpyAsync(A_d, A_h, mem_sizeA, hipMemcpyHostToDevice, stream));
  CU_CHECK(
      hipMemcpyAsync(B_d, B_h, mem_sizeB, hipMemcpyHostToDevice, stream));

  unsigned int gridRows = (dimsC.x + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int gridCols = (dimsC.y + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridSize(gridCols, gridRows); // blocks per grid
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE); // threads per block

  printf("[WARMING UP GPU]\n");
  // warm up runs 
  SlowMatMulKernel<<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, dimsA.x, dimsA.y, dimsB.y);
  MatMulKernel<BLOCK_SIZE><<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, dimsA.x, dimsA.y, dimsB.y);

  CU_CHECK(hipStreamSynchronize(stream));

  hipEvent_t start, stop;
  float elapsed;
  CU_CHECK(hipEventCreate(&start));
  CU_CHECK(hipEventCreate(&stop));

  printf("[RUNNING] SlowMatMul\n");
  CU_CHECK(hipEventRecord(start, stream));

  // <<<blocks in grid, block size (threads in block), dynamic shared mem,
  // gpu stream to run on>>>
  // NOTE: m = dimsA.x, n = dimsA.y, k = dimsB.y
  SlowMatMulKernel<<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, dimsA.x, dimsA.y, dimsB.y);

  CU_CHECK(hipEventRecord(stop, stream));
  CU_CHECK(hipEventSynchronize(stop));
  CU_CHECK(hipEventElapsedTime(&elapsed, start, stop));
  printf("[TIME] SlowMatMul completed in %.2fms.\n", elapsed);


  printf("[RUNNING] FastMatMul\n");
  CU_CHECK(hipEventRecord(start, stream));

  // <<<blocks in grid, block size (threads in block), dynamic shared mem,
  // gpu stream to run on>>>
  // NOTE: m = dimsA.x, n = dimsA.y, k = dimsB.y
  MatMulKernel<BLOCK_SIZE><<<gridSize, blockSize, 1, stream>>>(
      A_d, B_d, C_d, dimsA.x, dimsA.y, dimsB.y);

  CU_CHECK(hipEventRecord(stop, stream));
  CU_CHECK(hipEventSynchronize(stop));
  CU_CHECK(hipEventElapsedTime(&elapsed, start, stop));
  printf("[TIME] FastMatMul completed in %.2fms.\n", elapsed);

  CU_CHECK(hipEventDestroy(start));
  CU_CHECK(hipEventDestroy(stop));

  // wait host thread & error check
  CU_CHECK(hipGetLastError());
  CU_CHECK(hipStreamSynchronize(stream));

  // copy result back to host
  CU_CHECK(
      hipMemcpyAsync(C_h, C_d, mem_sizeC, hipMemcpyDeviceToHost, stream));
  CU_CHECK(hipStreamSynchronize(stream));

  if (DEBUG) {
    printf("\n[MAT A] (%dx%d):\n", dimsA.x, dimsA.y);
    printMat(A_h, size_A, dimsA.y);

    printf("\n[MAT B] (%dx%d):\n", dimsB.x, dimsB.y);
    printMat(B_h, size_B, dimsB.y);

    printf("\n[MAT C = A * B] (%dx%d):\n", dimsC.x, dimsC.y);
    printMat(C_h, size_C, dimsC.y);
  }

  if (CPU) {
    float* C = (float*)calloc(size_C, sizeof(float));

    auto s = std::chrono::high_resolution_clock::now();
    cpuMatMul(A_h, B_h, C, dimsA.x, dimsA.y, dimsB.y);
    auto e = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(e - s);
    printf("[TIME] CPU finished in %ldms.\n", duration.count());
    if (DEBUG) printMat(C, size_C, dimsC.y);

    free(C);
  }

  // free memory
  hipHostFree(A_h);
  hipHostFree(B_h);
  hipHostFree(C_h);
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  hipStreamDestroy(stream);
}

int main() {
  std::cout << "[CUDA] Launching matrix multiplication kernel...\n";

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  std::cout << "[DEVICE INFO]" << std::endl;
  std::cout << "DEVICE: " << prop.name << std::endl;
  std::cout << "SM COUNT: " << prop.multiProcessorCount << std::endl;
  std::cout << "MAX THREADS / BLOCK: " << prop.maxThreadsPerBlock << std::endl;
  std::cout << "WARP SIZE: " << prop.warpSize << std::endl;
  std::cout << "SHARED MEMORY / BLOCK: " << prop.sharedMemPerBlock / (1024) << "KB" << std::endl;
  std::cout << "GLOBAL MEMORY: " << prop.totalGlobalMem / (1024*1024*1024) << "GB" << std::endl;

  matMul();

  std::cout << "[END]" << std::endl;
  return 0;
}

void init_matmul(py::module_ &m) {
  m.def("matmul", &matMulNp,
    "Matrix multiplication: A @ B = C\n"
    "Args:\n"
    "  A: 1D array, shape (m*n) representing (m, n) matrix in row-major order\n"
    "  B: 1D array, shape (n*k) representing (n, k) matrix in row-major order\n"
    "  m, n, k: matrix dimensions\n"
    "Returns:\n"
    "  C: 2D array, shape (m, k)"
  );
}
