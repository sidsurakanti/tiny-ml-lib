#include "errors.cuh"
#include "matmul.cuh"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/pytypes.h>

namespace py = pybind11;

const int BLOCK_SIZE = 16;

py::capsule makeCapsule(void *ptr, bool isCudaPtr) {
  auto deleter =
      isCudaPtr ? [](void *p) { hipFree(p); } : [](void *p) { free(p); };

  return py::capsule(ptr, deleter);
}

typedef typename py::array_t<float, py::array::c_style | py::array::forcecast>
    py_ndarray_t;

void relu(py::capsule mat, int m, int n) {
  float *ptr = static_cast<float *>(mat.get_pointer());

  unsigned int gridRows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int gridCols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridDim(gridCols, gridRows);      // blocks per grid
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE); // threads per block

  ReluKernel<<<gridDim, blockDim>>>(ptr, m, n);
}

// we're gonna pass these capsules in from python and let them handle how they
// use the capsules we've init'd
// assume they're already on host and we just have to perform op
void linear(py::capsule X, py::capsule W, py::capsule b, py::capsule Y,
            int inputs, int outputs, int batch_size) {
  float *ptrX = static_cast<float *>(X.get_pointer()); // batchsize * inputs
  float *ptrW = static_cast<float *>(W.get_pointer()); // inputs * outputs
  float *ptrB = static_cast<float *>(b.get_pointer()); // 1 * outputs
  float *ptrY = static_cast<float *>(Y.get_pointer()); // batchsize * outputs

  int &m = batch_size;
  int &n = inputs;
  int &k = outputs;

  unsigned int gridRows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int gridCols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridDim(gridCols, gridRows);      // blocks per grid
  dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE); // threads per block

  MatMulKernel<BLOCK_SIZE><<<gridDim, blockDim>>>(ptrX, ptrW, ptrY, m, n, k);

  // wait host thread & error check
  CU_CHECK(hipGetLastError());      // launch errors
  CU_CHECK(hipDeviceSynchronize()); // kernel errors

  // output matrix is of size (batchsize, outputs) so we need a kernel size of
  // that
  unsigned int &vGridRows = gridRows;
  unsigned int vGridCols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 vGridDim(vGridCols, vGridRows);
  VecMatAddKernel<<<vGridDim, blockDim>>>(ptrB, ptrY, m, n);
}

py::array matMul(py_ndarray_t A, py_ndarray_t B, int m, int n, int k) {
  unsigned int size_A = m * n;
  unsigned int size_B = n * k;
  unsigned int size_C = m * k;

  unsigned int mem_sizeA = sizeof(float) * size_A;
  unsigned int mem_sizeB = sizeof(float) * size_B;
  unsigned int mem_sizeC = sizeof(float) * size_C;

  // unchecked is better than arr.request() for buf & then buf.ptr()
  // cus it auto throws when given more than 1d
  const float *A_h = A.unchecked<1>().data(0); // ptr to A[0]
  const float *B_h = B.unchecked<1>().data(0);
  float *C_h = (float *)calloc(m * k, sizeof(float));

  float *A_d, *B_d, *C_d;

  CU_CHECK(hipMalloc(&A_d, mem_sizeA));
  CU_CHECK(hipMalloc(&B_d, mem_sizeB));
  CU_CHECK(hipMalloc(&C_d, mem_sizeC));

  hipStream_t stream;
  hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  CU_CHECK(
      hipMemcpyAsync(A_d, A_h, mem_sizeA, hipMemcpyHostToDevice, stream));
  CU_CHECK(
      hipMemcpyAsync(B_d, B_h, mem_sizeB, hipMemcpyHostToDevice, stream));

  unsigned int gridRows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int gridCols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 gridSize(gridCols, gridRows);      // blocks per grid
  dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE); // threads per block

  MatMulKernel<BLOCK_SIZE>
      <<<gridSize, blockSize, 1, stream>>>(A_d, B_d, C_d, m, n, k);

  // wait host thread & error check
  CU_CHECK(hipGetLastError());
  CU_CHECK(hipStreamSynchronize(stream));

  // copy result back to host
  CU_CHECK(
      hipMemcpyAsync(C_h, C_d, mem_sizeC, hipMemcpyDeviceToHost, stream));
  CU_CHECK(hipStreamSynchronize(stream));

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
  hipStreamDestroy(stream);

  // wrap the c ptr in a capsule with a destructor so we can safely pass it
  // around we need to do this so when python destruct's its capsule obj we auto
  // free the memory for the c ptr
  py::capsule free_when_done = makeCapsule(C_h, false);
  py::array_t<float> result = py::array_t<float>({m, k}, // shape
                                                 C_h,    // data ptr
                                                 free_when_done);

  return result;
}

py::capsule initBuff(int m, int n) {
  int memsize = (m * n) * sizeof(float);
  float *ptr;

  CU_CHECK(hipMalloc(&ptr, memsize));
  CU_CHECK(hipMemset(ptr, 0, memsize));

  return makeCapsule(ptr, true);
}

py::capsule toGPU(py_ndarray_t obj, int size) {
  const float *dataPtr = obj.unchecked<1>().data(0); // ptr to weights
  int memsize = size * sizeof(float);
  float *ptr;

  CU_CHECK(hipMalloc(&ptr, memsize));
  CU_CHECK(hipMemcpy(ptr, dataPtr, memsize, hipMemcpyHostToDevice));

  return makeCapsule(ptr, true);
}

py::array toCPU(py::capsule cap, int m, int n) {
  float *ptr = static_cast<float *>(cap.get_pointer());
  int memsize = m * n * sizeof(float);
  float *retPtr = (float *)malloc(memsize);

  CU_CHECK(hipMemcpy(retPtr, ptr, memsize, hipMemcpyDeviceToHost));
  py::array_t<float> result = py::array_t<float>({m, n}, // shape
                                                 retPtr, // data ptr
                                                 makeCapsule(retPtr, false));
  return result;
}

void updateGpuMemory(py::capsule cap, int size) {
  float *ptrH = static_cast<float *>(cap.get_pointer());
  float *ptrD;

  CU_CHECK(
      hipMemcpy(ptrD, ptrH, size * sizeof(float), hipMemcpyHostToDevice));
}

auto initBuffers(py_ndarray_t W, int input_size, int output_size,
                 int batch_size) {

  const float *W_h = W.unchecked<1>().data(0); // ptr to weights

  // init W, b, dW, dB, C
  int sizeW = input_size * output_size;
  int sizeB = 1 * output_size;
  int sizeC = batch_size * output_size;

  int memsizeW = sizeW * sizeof(float);
  int memsizeB = sizeB * sizeof(float);
  int memsizeC = sizeC * sizeof(float);
  float *ptrW, *ptrB, *ptrC, *ptrdW, *ptrdB;

  CU_CHECK(hipMalloc(&ptrW, memsizeW));
  CU_CHECK(hipMalloc(&ptrB, memsizeB));
  CU_CHECK(hipMalloc(&ptrC, memsizeC));
  CU_CHECK(hipMalloc(&ptrdW, memsizeW));
  CU_CHECK(hipMalloc(&ptrdB, memsizeB));

  // copy over weight inits
  CU_CHECK(hipMemcpy(ptrW, W_h, memsizeW, hipMemcpyHostToDevice));

  // set all new init'd mat's to 0
  CU_CHECK(hipMemset(ptrdW, 0, memsizeW));
  CU_CHECK(hipMemset(ptrB, 0, memsizeB));
  CU_CHECK(hipMemset(ptrdB, 0, memsizeB));
  CU_CHECK(hipMemset(ptrC, 0, memsizeC));

  py::capsule w = makeCapsule(ptrW, true);
  py::capsule b = makeCapsule(ptrB, true);
  py::capsule c = makeCapsule(ptrC, true);
  py::capsule dw = makeCapsule(ptrdW, true);
  py::capsule db = makeCapsule(ptrdB, true);

  // access with capsule.get_pointer()
  return std::make_tuple(w, b, c, dw, db);
}

void init_matmul(py::module_ &m) {
  m.def("toGPU", &toGPU);
  m.def("toCPU", &toCPU);
  m.def("updateGpuMemory", &updateGpuMemory);
  m.def("initBuff", &initBuff);
  m.def("initBuffers", &initBuffers);
  m.def("linear", &linear);
  m.def("relu", &relu);
  m.def("matmul", &matMul,
        "Matrix multiplication: A @ B = C\n"
        "Args:\n"
        "  A: 1D array, shape (m*n) representing (m, n) matrix in row-major "
        "order\n"
        "  B: 1D array, shape (n*k) representing (n, k) matrix in row-major "
        "order\n"
        "  m, n, k: matrix dimensions\n"
        "Returns:\n"
        "  C: 2D array, shape (m, k)");
}
